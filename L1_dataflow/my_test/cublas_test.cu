#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <chrono>

#define CHECK_CUDA(call) \
    if((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(call) << std::endl; \
        exit(1); \
    }

#define CHECK_CUBLAS(call) \
    if((call) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    }

void initialize_matrix(half *matrix, int rows, int cols, half value) {
    half *host_matrix = new half[rows * cols];
    for (int i = 0; i < rows * cols; ++i) {
        host_matrix[i] = value;
    }
    CHECK_CUDA(hipMemcpy(matrix, host_matrix, rows * cols * sizeof(half), hipMemcpyHostToDevice));
    delete[] host_matrix;
}

void print_first_element(half *matrix) {
    half host_value;
    CHECK_CUDA(hipMemcpy(&host_value, matrix, sizeof(half), hipMemcpyDeviceToHost));
    std::cout << "First element: " << __half2float(host_value) << std::endl;
}

void handle_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "hipblasGemmEx failed with error code: " << status << std::endl;
        switch(status) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                std::cerr << "CUBLAS_STATUS_NOT_INITIALIZED" << std::endl;
                break;
            case HIPBLAS_STATUS_ALLOC_FAILED:
                std::cerr << "CUBLAS_STATUS_ALLOC_FAILED" << std::endl;
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                std::cerr << "CUBLAS_STATUS_INVALID_VALUE" << std::endl;
                break;
            case HIPBLAS_STATUS_ARCH_MISMATCH:
                std::cerr << "CUBLAS_STATUS_ARCH_MISMATCH" << std::endl;
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                std::cerr << "CUBLAS_STATUS_MAPPING_ERROR" << std::endl;
                break;
            case HIPBLAS_STATUS_EXECUTION_FAILED:
                std::cerr << "CUBLAS_STATUS_EXECUTION_FAILED" << std::endl;
                break;
            case HIPBLAS_STATUS_INTERNAL_ERROR:
                std::cerr << "CUBLAS_STATUS_INTERNAL_ERROR" << std::endl;
                break;
            case HIPBLAS_STATUS_NOT_SUPPORTED:
                std::cerr << "CUBLAS_STATUS_NOT_SUPPORTED" << std::endl;
                break;
            case HIPBLAS_STATUS_UNKNOWN:
                std::cerr << "CUBLAS_STATUS_LICENSE_ERROR" << std::endl;
                break;
            default:
                std::cerr << "Unknown cublas status" << std::endl;
        }
        exit(1);
    }
}

int main() {
    const int M = 4096;
    const int N = 20480;
    const int K = 5120;
    const int iterations = 1000;

    // Allocate device memory
    half *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void**)&d_A, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc((void**)&d_B, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc((void**)&d_C, M * N * sizeof(half)));

    // Initialize matrices
    initialize_matrix(d_A, M, K, __float2half(1.0f));
    initialize_matrix(d_B, K, N, __float2half(1.0f));
    initialize_matrix(d_C, M, N, __float2half(0.0f));

    // Initialize cuBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set cuBLAS to use Tensor Cores
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // Define scaling factors
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    // A is M x K
    // B is K x N
    // C is M x N

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    // A is M x K
    // B is K x N
    // C is M x N

    for (int i = 0; i < iterations; ++i) {
        hipblasStatus_t status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  N, M, K,
                                  &alpha,
                                  d_B, HIP_R_16F, N,
                                  d_A, HIP_R_16F, K,
                                  &beta,
                                  d_C, HIP_R_16F, N,
                                  HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));

    // Wait for the stop event to complete
    CHECK_CUDA(hipEventSynchronize(stop));

    // Calculate the elapsed time
    float elapsed_time_ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time_ms, start, stop));

    float average_time_ms = elapsed_time_ms / iterations;

    // Print the first element of the result matrix
    print_first_element(d_C);

    // Print the average execution time
    std::cout << "Average execution time: " << average_time_ms << " ms" << std::endl;

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));

    std::cout << "Matrix multiplication completed successfully!" << std::endl;
    return 0;
}
