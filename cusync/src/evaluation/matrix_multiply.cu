#include <iostream>
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define CHECK_CUDA(call) \
    if((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::exit(EXIT_FAILURE); \
    }

#define CHECK_CUBLAS(call) \
    if((call) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::exit(EXIT_FAILURE); \
    }

void initializeMatrix(half *matrix, int size, half value) {
    for (int i = 0; i < size; i++) {
        matrix[i] = value;
    }
}

int main() {
    const int m = 512;
    const int n = 512;
    const int k = 512;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    half alpha = __float2half(1.0f);
    half beta = __float2half(0.0f);

    // Allocate memory for matrices on the host
    std::vector<half> h_A(m * k, __float2half(0.1f));
    std::vector<half> h_B(k * n, __float2half(0.1f));
    std::vector<half> h_C(m * n, __float2half(0.0f));

    // Allocate memory for matrices on the device
    half *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, m * k * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B, k * n * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C, m * n * sizeof(half)));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), m * k * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), k * n * sizeof(half), hipMemcpyHostToDevice));

    // Perform matrix multiplication C = A * B
    CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                              n, m, k,
                              &alpha,
                              d_B, HIP_R_16F, n,
                              d_A, HIP_R_16F, k,
                              &beta,
                              d_C, HIP_R_16F, n,
                              HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, m * n * sizeof(half), hipMemcpyDeviceToHost));

    // Print the first 10 elements of matrix C
    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << __half2float(h_C[i]) << std::endl;
    }

    // Free device memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    // Destroy the cuBLAS handle
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
