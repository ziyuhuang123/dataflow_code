#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

//<OPTIMIZATIONS>
#define MLP_GPT3
#undef AVOID_CUSTOM_ORDER
#undef AVOID_WAIT_KERNEL
#undef REORDER_TILE_LOADS
#undef NO_ATOMIC_ADD
#define ROWSYNC
#define EVAL_TILE_SIZES

//</OPTIMIZATIONS>

// #define LLAMA

#if defined(TILESYNC)
#if !defined(MLP_LLAMA)
  #define NO_ATOMIC_ADD
#else
  #undef NO_ATOMIC_ADD
#endif
#define REORDER_TILE_LOADS
#endif

// #define AVOID_CUSTOM_ORDER
// #define AVOID_WAIT_KERNEL

// #if defined(TILESYNC) || defined(TILEBATCH)
// #endif 

#include<cusync/cusync.h>

#include "cutlass/cusync-cutlass/include/cutlass/gemm/kernel/default_cusyncgemm.h"
#include "/home/zyhuang/temp_can/dataflow_code/cusync/src/include/cutlass/nvidia-cutlass/include/cutlass/gemm/device/default_gemm_configuration.h"
#include "/home/zyhuang/temp_can/dataflow_code/cusync/src/include/cutlass/nvidia-cutlass/include/cutlass/arch/mma.h"
#include "/home/zyhuang/temp_can/dataflow_code/cusync/src/include/cutlass/nvidia-cutlass/include/cutlass/arch/arch.h"
#include "/home/zyhuang/temp_can/dataflow_code/cusync/src/include/cutlass/nvidia-cutlass/include/cutlass/gemm/gemm.h"
#include "/home/zyhuang/temp_can/dataflow_code/cusync/src/include/cutlass/nvidia-cutlass/include/cutlass/layout/permute.h"


using namespace cusync;

const uint Opts = 
#ifdef AVOID_CUSTOM_ORDER
  Optimizations::AvoidCustomOrder |
#endif
#ifdef AVOID_WAIT_KERNEL
  Optimizations::AvoidWaitKernel  |
#endif
#ifdef NO_ATOMIC_ADD
  Optimizations::NoAtomicAdd      |
#endif
#ifdef REORDER_TILE_LOADS
  Optimizations::ReorderTileLoads |
#endif
  Optimizations::NoOptimization;

#include "common.h"

//Tile sizes of all GeMMs
using ShapeThreadBlock1 = cutlass::gemm::GemmShape<256, 128, 32>;
using ShapeWarp1 = cutlass::gemm::GemmShape<64, 64, 32>;

using ShapeThreadBlock2 = cutlass::gemm::GemmShape<256, 128, 32>;
using ShapeWarp2 = cutlass::gemm::GemmShape<64, 64, 32>;

const int NumStages1 = 4;
const int NumStages2 = 4;



#define XSTR(x) STR(x)
#define STR(x) #x

using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  
using SmArch = cutlass::arch::Sm80;


using ProdCuStage   = CuStage<TransposeXYOrder, NoSync,  RowSync<ShapeThreadBlock1::kM>, Opts>;
using ConsCuStage   = CuStage<TransposeXYOrder, RowSync<ShapeThreadBlock1::kM>, NoSync,  Opts>;
using Sync = RowSync<ShapeThreadBlock1::kM>;

const uint GLURowTile = 8;


//Element types of A, B, and C
using ElementAccumulator = float;
using ElementInputA = cutlass::half_t;
using ElementInputB = cutlass::half_t;
using ElementOutput = cutlass::half_t;
using ElementComputeEpilogue = cutlass::half_t;

//All matrices are in RowMajor
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::RowMajor;
using LayoutOutput = cutlass::layout::RowMajor;

//Use FP-16 Tensor Cores
using MMAOp = cutlass::arch::OpClassTensorOp;

#ifdef EVAL_TILE_SIZES
  //During evaluation apply correct epilogue op
  #ifdef MLP_LLAMA
    //First GeMM in LLaMA does not apply SwiGLU but is done in 
    //another kernel
    using EpilogueOp1 = cutlass::epilogue::thread::LinearCombination<
  #elif defined(MLP_GPT3)
    //First GeMM in MLP is fused with GELU
    using EpilogueOp1 = cutlass::epilogue::thread::LinearCombinationGELU<
  #endif
#else
  //For correctness check no need to appy any epilogue
  using EpilogueOp1 = cutlass::epilogue::thread::LinearCombination<
#endif
    ElementOutput,                                        
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementComputeEpilogue>;
    // cutlass::epilogue::thread::ScaleType::NoBetaScaling>;

//Second GeMM in MLP performs no extra fused computations 
using EpilogueOp2 = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        
    128 / cutlass::sizeof_bits<ElementOutput>::value,     
    ElementAccumulator,
    ElementComputeEpilogue>;

template<typename EpilogueOp, typename ShapeThreadBlock, typename ShapeWarp, int NumStages, bool splitK>
class BaseMLPGemm : public cutlass::gemm::device::Gemm<ElementInputA, LayoutInputA, 
                                                       ElementInputB, LayoutInputB,
                                                       ElementOutput, LayoutOutput,
                                                       ElementAccumulator, MMAOp,
                                                       SmArch, ShapeThreadBlock,
                                                       ShapeWarp, ShapeMMAOp,
                                                       EpilogueOp, 
                                                       cutlass::gemm::threadblock::CuSyncGemmHorizontalThreadblockSwizzle,
                                                       NumStages, 8, 8, splitK> {};
// Baseline GeMMs
using Gemm1 = BaseMLPGemm<EpilogueOp1, ShapeThreadBlock1, ShapeWarp1, NumStages1, false>;
using Gemm2 = BaseMLPGemm<EpilogueOp2, ShapeThreadBlock2, ShapeWarp2, NumStages2, false>;

//Baseline GeMMs with SplitK enabled
using GemmSplitK1 = BaseMLPGemm<EpilogueOp1, ShapeThreadBlock1, ShapeWarp1, NumStages1, true>;
using GemmSplitK2 = BaseMLPGemm<EpilogueOp2, ShapeThreadBlock2, ShapeWarp2, NumStages2, true>;

//CuSync GeMMs
using CuSyncGeMMSwizzle = cutlass::gemm::threadblock::CuSyncGemmHorizontalThreadblockSwizzle;
template<typename CuStage, typename EpilogueOp, typename ShapeThreadBlock, typename ShapeWarp, int NumStages, bool splitK>
class CuSyncMLPGemm : public cutlass::gemm::device::CuSyncGemm<CuStage,
                                                               ElementInputA, LayoutInputA, 
                                                               ElementInputB, LayoutInputB,
                                                               ElementOutput, LayoutOutput,
                                                               ElementAccumulator, MMAOp,
                                                               SmArch, ShapeThreadBlock,
                                                               ShapeWarp, ShapeMMAOp,
                                                               EpilogueOp, 
                                                               CuSyncGeMMSwizzle,
                                                               NumStages, 8, 8, splitK> {};

using CuSyncGemm1 = CuSyncMLPGemm<ProdCuStage, EpilogueOp1, ShapeThreadBlock1, ShapeWarp1, NumStages1, false>;
using CuSyncGemm2 = CuSyncMLPGemm<ConsCuStage, EpilogueOp2, ShapeThreadBlock2, ShapeWarp2, NumStages2, false>;

using CuSyncGemmSplitK1 = CuSyncMLPGemm<ProdCuStage, EpilogueOp1, ShapeThreadBlock1, ShapeWarp1, NumStages1, true>;
using CuSyncGemmSplitK2 = CuSyncMLPGemm<ConsCuStage, EpilogueOp2, ShapeThreadBlock2, ShapeWarp2, NumStages2, true>;

using HostTensor = cutlass::HostTensor<ElementInputA, LayoutInputA>;

enum MLPType {
  GPT3,
  LLaMa    
};

struct MLPParameters {
  HostTensor x; //[B, H]
  HostTensor w1; //[H, 4H/8] in GPT-3
  //xw1 = GeLU(x * w1)
  HostTensor xw1; //[B, 4 H / 8]
  HostTensor w2; //[4H/8, H] in GPT-3 and [H/3, H] in LLaMa
  //xw12 = xw1 * w2
  HostTensor xw12; //[B, H]

  //For LLaMa only
  HostTensor vw1; //[B, 2*H/3] in LLAMA
  HostTensor xvw1; //[B, 2*H/3] in LLaMa
  HostTensor glu; //[B, H/3] in LLaMa

  HostTensor ref_xw1;
  HostTensor ref_xw12;

  //For LLaMa only
  HostTensor ref_xv;

  bool checkResults;

  cutlass::gemm::GemmCoord gemm_size1;
  cutlass::gemm::GemmCoord gemm_size2;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;

  std::string model;

  MLPParameters(std::string model_, uint batch, bool check) {
    alpha = ElementComputeEpilogue(1.0);
    beta = ElementComputeEpilogue(0.0);
    model = model_;

    if (model == "gpt3") {
      gemm_size1 = cutlass::gemm::GemmCoord(batch, 4*12288/8, 12288);
      gemm_size2 = cutlass::gemm::GemmCoord(batch, 12288, 4*12288/8);
    } else if (model=="llama") {
      int H = 8192;
      int d = ((H/3 + 127)/128)*128;
      gemm_size1 = cutlass::gemm::GemmCoord(batch, 2*d, H);
      gemm_size2 = cutlass::gemm::GemmCoord(batch, H, d);
    }
    std::cout << "GeMM 1 Size: " << gemm_size1.m() << ", " << 
      gemm_size1.n() << ", " << gemm_size1.k() << std::endl;
    std::cout << "GeMM 2 Size: " << gemm_size2.m() << ", " << 
      gemm_size2.n() << ", " << gemm_size2.k() << std::endl;
    
    x = HostTensor(gemm_size1.mk());
    w1 = HostTensor(gemm_size1.kn());
    xw1 = HostTensor(gemm_size1.mn());
    w2 = HostTensor(gemm_size2.kn());
    xw12 = HostTensor(gemm_size2.mn());
    ref_xw1 = HostTensor(gemm_size1.mn());
    ref_xw12 = HostTensor(gemm_size2.mn());

    if (model == "llama") {
      xvw1 = HostTensor(gemm_size1.mn());
      vw1 = HostTensor(gemm_size1.kn());
      glu = HostTensor(gemm_size2.mk());
      ref_xv = HostTensor(gemm_size1.mn());
    }
    checkResults = check;
  }

  void initIns() {
    if (checkResults) {
      ElementOutput values[5] = {ElementOutput(0.05), ElementOutput(0.3),
                                 ElementOutput(0.1), ElementOutput(0.06),
                                 ElementOutput(0.04)};
      memset_random(x.host_data(), 5, values, x.size());
      memset_random(w1.host_data(), 5, values, w1.size());
      memset_random2(w2.host_data(), ElementOutput(0.01), ElementOutput(0.05), w2.size());
      if (model == "llama") {
        memset_random2(vw1.host_data(), ElementOutput(0.01), ElementOutput(0.2), vw1.size());
      }
    } else {
      cutlass::reference::host::TensorFill(x.host_view(), ElementOutput(0.05));
      cutlass::reference::host::TensorFill(w1.host_view(), ElementOutput(0.5));
      cutlass::reference::host::TensorFill(w2.host_view(), ElementOutput(0.01));
      if (model == "llama") {
        cutlass::reference::host::TensorFill(vw1.host_view(), ElementOutput(0.5));
      }
    }
    // Copy data from host to GPU
    x.sync_device();
    w1.sync_device();
    w2.sync_device();
    if (model == "llama") {
      vw1.sync_device();
    }
  }
  
  void initOuts() {
    cutlass::reference::host::TensorFill(xw1.host_view());
    cutlass::reference::host::TensorFill(xw12.host_view());
      
    xw1.sync_device();
    xw12.sync_device();
    if (model == "llama") {
      cutlass::reference::host::TensorFill(xvw1.host_view());
      xvw1.sync_device();
      cutlass::reference::host::TensorFill(glu.host_view());
      glu.sync_device();
    }
  }

  void initRefs() {
    cutlass::reference::host::TensorFill(ref_xw12.host_view());
    cutlass::reference::host::TensorFill(ref_xw1.host_view());

    ref_xw12.sync_device();
    ref_xw1.sync_device();
    if (model == "llama") {
      cutlass::reference::host::TensorFill(ref_xv.host_view());
      ref_xv.sync_device(); 
    }
  }

  bool isGPT3() {return model == "gpt3";}
  bool isLLaMa() {return model == "llama";}
};

/** Reference MLP for correctness check **/
hipError_t referenceMLP(MLPParameters& mlpParams) {
  ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size1.m(), 
                                                mlpParams.gemm_size1.n(), 
                                                mlpParams.gemm_size1.k(),
                                                mlpParams.x.device_data(), 
                                                mlpParams.w1.device_data(), 
                                                mlpParams.ref_xw1.host_data());
  CUDA_CHECK(hipMemcpy(mlpParams.ref_xw1.device_data(), mlpParams.ref_xw1.host_data(), 
             sizeof(ElementOutput) * mlpParams.ref_xw1.size(), hipMemcpyHostToDevice));
  
  if (mlpParams.isLLaMa()) {
    printf("check not supported in llama\n");
    return hipSuccess;
    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size1.m(), 
                                                  mlpParams.gemm_size1.n(), 
                                                  mlpParams.gemm_size1.k(),
                                                  mlpParams.x.device_data(), 
                                                  mlpParams.vw1.device_data(), 
                                                  mlpParams.ref_xv.host_data());
    //Compute XW1 (dot) XV
    for (int b = 0; b < mlpParams.gemm_size1.m(); b++) {
      for (int n = 0; n < mlpParams.gemm_size1.n(); n++) {
        uint index = b * mlpParams.gemm_size1.n() + n;
        mlpParams.ref_xv.host_data()[index] = mlpParams.ref_xw1.host_data()[index] * 
                                              mlpParams.ref_xv.host_data()[index];
      }
    }

    mlpParams.ref_xv.sync_device();

    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size2.m(),
                                                  mlpParams.gemm_size2.n(),
                                                  mlpParams.gemm_size2.k(), 
                                                  mlpParams.ref_xv.device_data(),
                                                  mlpParams.w2.device_data(), 
                                                  mlpParams.ref_xw12.host_data());
  } else {
    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size2.m(),
                                                  mlpParams.gemm_size2.n(),
                                                  mlpParams.gemm_size2.k(), 
                                                  mlpParams.ref_xw1.device_data(),
                                                  mlpParams.w2.device_data(), 
                                                  mlpParams.ref_xw12.host_data());
  }

  return hipSuccess;
}

hipError_t checkMLPResults(MLPParameters& mlpParams) {
  ElementOutput* hostC = new ElementOutput[mlpParams.ref_xw1.size()];
  CUDA_CHECK(hipMemcpy(hostC, mlpParams.xw1.device_data(), 
                        mlpParams.xw1.size() * sizeof(ElementOutput), 
                        hipMemcpyDeviceToHost));
  printf("Checking first GeMM\n");
  bool eq = equals(mlpParams.ref_xw1.size(), mlpParams.ref_xw1.host_data(), hostC, 1e-1f);
  if (eq == false) {
    printf("First GeMM not correct\n");
    return hipErrorUnknown;
  }
  printf("First GeMM passed\n");
  ElementOutput* hostE = new ElementOutput[mlpParams.ref_xw12.size()];
  CUDA_CHECK(hipMemcpy(hostE, mlpParams.xw12.device_data(), 
                        mlpParams.xw12.size() * sizeof(ElementOutput), 
                        hipMemcpyDeviceToHost));
  //For LLaMa not checking XV
  printf("Checking second GeMM\n");
  eq = equals(mlpParams.ref_xw12.size(), mlpParams.ref_xw12.host_data(), hostE, 1e-1f);
  if (eq == false) {
    printf("Second GeMM not correct \n");
    return hipErrorUnknown;
  }

  printf("Second GeMM passed\n");

  return hipSuccess;
}

/*GPT3 Baseline MLP*/
template<typename GemmTy1, typename GemmTy2>
hipError_t runBaselineGPT3(int split_k1, int split_k2, 
                            MLPParameters& mlpParams,
                            hipStream_t stream,
                            double& execTime, double& matmul1Time, double& softmaxTime, double& matmul2Time,
                            int iters = 100) {
  //Setup first GeMM
  typename GemmTy1::Arguments args1 {
    mlpParams.gemm_size1,
    mlpParams.x.device_ref(), 
    mlpParams.w1.device_ref(),
    mlpParams.xw1.device_ref(),
    mlpParams.xw1.device_ref(),
    {mlpParams.alpha, mlpParams.beta},
    split_k1};

  size_t workspace_size = GemmTy1::get_workspace_size(args1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  GemmTy1 gemm_op1;
  cutlass::Status status = gemm_op1.can_implement(args1);
  CUTLASS_CHECK(status);
  status = gemm_op1.initialize(args1, workspace1.get());
  CUTLASS_CHECK(status);

  //Setup Second GeMM
  typename GemmTy2::Arguments args2{ 
    mlpParams.gemm_size2, 
    mlpParams.xw1.device_ref(), 
    mlpParams.w2.device_ref(), 
    mlpParams.xw12.device_ref(), 
    mlpParams.xw12.device_ref(), 
    {mlpParams.alpha, mlpParams.beta},         
    split_k2};
  
  GemmTy2 gemm_op2;
  workspace_size = GemmTy2::get_workspace_size(args2);
  cutlass::device_memory::allocation<uint8_t> workspace2(workspace_size);
  status = gemm_op2.can_implement(args2);
  CUTLASS_CHECK(status);
  status = gemm_op2.initialize(args2, workspace2.get());
  CUTLASS_CHECK(status);
  
  execTime = 0;
  hipStream_t stream2;
  CUDA_CHECK(hipStreamCreate(&stream2));

  hipEvent_t start, end, middle;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));
  CUDA_CHECK(hipEventCreate(&middle));

  //Run kernels
  for (int r = 0; r < iters; r++) {    
    CUDA_CHECK(hipEventRecord(start, stream));
    status = gemm_op1(stream);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipEventRecord(middle, stream));

    status = gemm_op2(stream);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipEventRecord(end, stream));
    CUDA_CHECK(hipEventSynchronize(end));

    float iterMatMul1 = 0;
    CUDA_CHECK(hipEventElapsedTime(&iterMatMul1, start, middle));
    matmul1Time += iterMatMul1;
    float iterMatMul2 = 0;
    CUDA_CHECK(hipEventElapsedTime(&iterMatMul2, middle, end));
    matmul2Time += iterMatMul2;

    float end_to_start = 0;
    CUDA_CHECK(hipEventElapsedTime(&end_to_start, start, end));

    if (iters == 20)
      printf("{\"Total\": %lf, \"matmul1Time\": %lf, \"matmul2Time\": %lf}\n", 
             end_to_start * 1000.0f, iterMatMul1*1000.0f, iterMatMul2*1000.0f);
    execTime += end_to_start * 1000.0f;
  }

  return hipSuccess;
}

hipError_t runBaselineGPT3(int split_k1, int split_k2, 
                        MLPParameters& mlpParams,
                        hipStream_t stream,
                        double& execTime,
                        double& matmul1Time,
                        double& softmaxTime,
                        double& matmul2Time,
                        int iters = 100) {
  hipError_t result;
  execTime = 0;
  matmul1Time = 0;
  softmaxTime = 0;
  matmul2Time = 0;
  if (split_k1 == 1 && split_k2 == 1) {
    result = runBaselineGPT3<Gemm1, Gemm2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runBaselineGPT3<GemmSplitK1, Gemm2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runBaselineGPT3<Gemm1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else {
    result = runBaselineGPT3<GemmSplitK1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  }

  return result;
}

/*LLaMA Baseline MLP*/
template<typename T, uint H3>
__global__ void gluKernel(T* xvw1, T* glu) {
  int ROW = blockIdx.x;

  for (int i = threadIdx.x; i < H3; i += blockDim.x) {
    float xw1 = xvw1[ROW * (2 * H3) + i];
    float xv =  xvw1[ROW * (2 * H3) + i + H3];
    glu[ROW * H3 + i] = xw1 * xv;
  }
}

template <typename Operator>
__device__
void GEMMdeviceFunction(typename Operator::Params& params) {
  // Dynamic shared memory base pointer
  extern __shared__ int SharedStorageBase[];

  // Declare pointer to dynamic shared memory.
  typename Operator::SharedStorage *shared_storage =
      reinterpret_cast<typename Operator::SharedStorage *>(SharedStorageBase);

  Operator op;
  op(params, *shared_storage);
}

/// Generic CUTLASS kernel template.
template <typename Operator>
__global__
void AllKernel(typename Operator::Params params) {
  // 调用device function来执行op，并显式传递模板参数
  GEMMdeviceFunction<Operator>(params);
}



/*CuSync GPT-3 MLP*/
template<typename GemmTy1, typename GemmTy2>
hipError_t runCuSyncGPT3(int split_k1, int split_k2,
                          MLPParameters& mlpParams,
                          ProdCuStage& prod, ConsCuStage& cons,
                          hipStream_t producer_stream, 
                          hipStream_t consumer_stream,
                          double& execTime,
                          int iters = 100) {
  typename GemmTy1::Arguments args1{prod,
                                     mlpParams.gemm_size1,
                                     mlpParams.x.device_ref(),
                                     mlpParams.w1.device_ref(),
                                     mlpParams.xw1.device_ref(),
                                     mlpParams.xw1.device_ref(),
                                     {mlpParams.alpha, mlpParams.beta},         
                                     split_k1};
  GemmTy1 gemm_op1;
  size_t workspace_size = GemmTy1::get_workspace_size(args1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  cutlass::Status status = gemm_op1.can_implement(args1);
  CUTLASS_CHECK(status);
  status = gemm_op1.initialize(args1, workspace1.get());
  CUTLASS_CHECK(status);

  typename GemmTy2::Arguments args2{cons,
                                    mlpParams.gemm_size2,  
                                    mlpParams.xw1.device_ref(),
                                    mlpParams.w2.device_ref(),
                                    mlpParams.xw12.device_ref(),
                                    mlpParams.xw12.device_ref(),
                                    {mlpParams.alpha, mlpParams.beta},
                                    split_k2};

  GemmTy2 gemm_op2;
  workspace_size = GemmTy2::get_workspace_size(args2);
  cutlass::device_memory::allocation<uint8_t> workspace2(workspace_size);
  status = gemm_op2.can_implement(args2);
  CUTLASS_CHECK(status);
  status = gemm_op2.initialize(args2, workspace2.get());
  CUTLASS_CHECK(status);

  execTime = 0;
  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  for (int r = 0; r < iters; r++) {
    CUDA_CHECK(hipEventRecord(start, producer_stream));
    status = gemm_op1.run(true, NULL, producer_stream);

    /// Operator class tag
    // using OperatorClass_ = cutlass::arch::OpClassTensorOp; // 这里一开始总是报错，关键是要加上cutlass::arch，以及要include相应文件，就去文件里找，这个对应的是nvcutlass底下arch的mma.h，所以include就好。(可以直接用绝对路径)。另一种方法，因为wmma.h里面也include了同一个mma.h。其实include wmma.h也是可以的。（后来发现前面又MMAOp是一样的内容）

    // Access granularity of A matrix in units of elements
    int AlignmentA =
        cutlass::gemm::device::DefaultGemmConfiguration<MMAOp, SmArch, ElementInputA, ElementInputB,
                                 ElementOutput, ElementAccumulator>::kAlignmentA;

    // Access granularity of B matrix in units of elements
    int AlignmentB =
        cutlass::gemm::device::DefaultGemmConfiguration<MMAOp, SmArch, ElementInputA, ElementInputB,
                                 ElementOutput, ElementAccumulator>::kAlignmentB;
    static int const kAlignmentA = AlignmentA;
    static int const kAlignmentB = AlignmentB;

    using Operator_ =
        cutlass::gemm::device::DefaultGemmConfiguration<MMAOp, SmArch, ElementInputA, ElementInputB,
                                 ElementOutput, ElementAccumulator>::Operator;

    auto SharedClearOption = cutlass::gemm::SharedMemoryClearOption::kNone;
    using PermuteDLayout = cutlass::layout::NoPermute;

    using GemmKernel = typename cutlass::gemm::kernel::DefaultCuSyncGemm<ProdCuStage,
                                       ElementInputA, 
                                       LayoutInputA,
                                       kAlignmentA, 
                                       ElementInputB, 
                                       LayoutInputB,
                                       kAlignmentB,
                                       ElementOutput, 
                                       LayoutOutput,
                                       ElementAccumulator, 
                                       MMAOp,
                                       SmArch, 
                                       ShapeThreadBlock1,
                                       ShapeWarp1, 
                                       ShapeMMAOp,
                                       EpilogueOp1, 
                                       CuSyncGeMMSwizzle,
                                       NumStages1, 
                                       false, 
                                       Operator_, 
                                       SharedClearOption,
                                       false,
                                       false,
                                       false,
                                       PermuteDLayout
                                       >::GemmKernel;

  //   kStages,
  //   kSplitKSerial,
  //   Operator,
  //   SharedMemoryClearOption::kNone,
  //   GatherA,
  //   GatherB,
  //   ScatterD,
  //   PermuteDLayout

  // using Operator = Operator_;
  // static int const kStages = Stages;
  // static int const kAlignmentA = AlignmentA;
  // static int const kAlignmentB = AlignmentB;
  // static int const kAlignmentC = EpilogueOutputOp::kCount;
  // static bool const kSplitKSerial = SplitKSerial;
  //   /// If true, kernel supports split-K with serial reduction
  //   bool SplitKSerial = false,
  //   /// Operation performed by GEMM
  //   typename Operator_ = typename DefaultGemmConfiguration<
  //       OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
  //       ElementAccumulator_>::Operator,
  //   /// Gather operand A by using an index array
  //   bool GatherA = false,
  //   /// Gather operand B by using an index array
  //   bool GatherB = false,
  //   /// Scatter result D by using an index array
  //   bool ScatterD = false,
  //   /// Permute result D
  //   typename PermuteDLayout = layout::NoPermute



    CuSyncGeMMSwizzle cuSyncGeMMSwizzle;
    // dim3 grid = cuSyncGeMMSwizzle.get_grid_shape(params_.grid_tiled_shape);
    // dim3 block(GemmKernel::kThreadCount, 1, 1);
    // int smem_size = 100 << 10;
    // AllKernel<GemmKernel><<<grid, block, smem_size>>>(args1);



    CUTLASS_CHECK(status);
    // CUDA_CHECK(hipDeviceSynchronize());
    // CUDA_CHECK(hipDeviceSynchronize());
    // prod.invokeWaitKernel(consumer_stream);  
    // // CUDA_CHECK(hipDeviceSynchronize());
    // status = gemm_op2.run(true, NULL, consumer_stream);
    // CUDA_CHECK(hipEventRecord(end, consumer_stream));
    // CUDA_CHECK(hipEventSynchronize(end));
    // CUTLASS_CHECK(status);
    // float time_ms = 0;
    // CUDA_CHECK(hipEventElapsedTime(&time_ms, start, end));
    
    // if (iters > 10)
    //   printf("{\"Total\": %lf}\n",time_ms*1000.0f);
    // execTime += time_ms*1000.0f;
    // prod.incrementIter();
    // cons.incrementIter();
    // gemm_op2.params_.custage.incrementIter();
    // gemm_op1.params_.custage.incrementIter();
  }

  return hipSuccess;
}

hipError_t runCuSyncGPT3(int split_k1, int split_k2, MLPParameters& mlpParams,
                          ProdCuStage& prod, ConsCuStage& cons,
                          hipStream_t producer_stream, hipStream_t consumer_stream,
                          double& execTime, int iters = 100) {
  hipError_t result;
  execTime = 0;

  if (split_k1 == 1 && split_k2 == 1) {
    result = runCuSyncGPT3<CuSyncGemm1, CuSyncGemm2>(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, execTime, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runCuSyncGPT3<CuSyncGemmSplitK1, CuSyncGemm2>(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, execTime, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runCuSyncGPT3<CuSyncGemm1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, execTime, iters);
  } else {
    result = runCuSyncGPT3<CuSyncGemmSplitK1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, execTime, iters);
  }

  return result;
}

int run(int argc, char* argv[]) {
  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  // if (props.major != 7) {
  //   std::cerr << "Volta Tensor Ops must be run on a machine"
  //             << "with compute capability of 70, 72, or 75."
  //             << std::endl;
  //   return 0;
  // }
  
  const uint NUM_ARGS = 6;
  std::string argNames[NUM_ARGS] = {"--model", "--batch", "--check", "--split-k1", "--split-k2", "--policy"};
  std::string argHelp[NUM_ARGS] = {"GPT3 or LLaMa", "Batch size", "Check results", 
                                   "Split K for first GeMM", "Split K for second GeMM",
                                   "Policy to execute"};
  
  if (argc < NUM_ARGS+1) {
    std::cout << "usage: " << std::endl
              << argNames[0] << " gpt3|llama " << argHelp[0] << std::endl 
              << argNames[1] << " <int>" << argHelp[1] << std::endl
              << argNames[2] << " true|false" << argHelp[2] << std::endl
              << argNames[3] << " <int> " << argHelp[3] << std::endl
              << argNames[4] << " <int> " << argHelp[4] << std::endl
              << argNames[5] << " baseline|cusync" << argHelp[5] << std::endl;
    return 0;
  }

  std::string model = "", policy = "";
  uint batch = 0;
  bool doChecking = false;
  uint split_k1 = 1;
  uint split_k2 = 1;

  for (int i = 1; i < argc; ++i) {
    std::string arg = std::string(argv[i]);
    if (arg.find(argNames[0]) == 0) {
      model = std::string(argv[i+1]);
      i = i + 1;
    } else if (arg.find(argNames[1]) == 0) {
      std::stringstream ss(argv[i+1]);
      ss >> batch;
      i = i + 1;
    } else if (arg.find(argNames[2]) == 0) {
      std::string val = std::string(argv[i+1]);
      if (val == "true") {
        doChecking = true;
      } else if (val == "false") {
        doChecking = false;
      } else {
        std::cout << "Invalid value for check " << val << std::endl;
      }
      i = i + 1;
    } else if (arg.find(argNames[3]) == 0) {
      split_k1 = atoi(argv[i+1]);
      i=i+1;
    } else if (arg.find(argNames[4]) == 0) {
      split_k2 = atoi(argv[i+1]);
      i=i+1;
    } else if (arg.find(argNames[5]) == 0) {
      policy = std::string(argv[i+1]);
      i=i+1;
    }
  }

  if (model == "" || batch == 0) {
    std::cout<<"invalid model or batch" <<std::endl;
    return 0;
  }
    
  std::cout << "model=" << model << " batch=" << batch << " check="<<doChecking << " policy= " << policy << std::endl;

  hipStream_t producer_stream;
  hipStream_t producer_stream2;
  hipStream_t consumer_stream;
  CUDA_CHECK(hipStreamCreate(&producer_stream));
  CUDA_CHECK(hipStreamCreate(&producer_stream2));
  CUDA_CHECK(hipStreamCreate(&consumer_stream));

  MLPParameters mlpParams(model, batch, doChecking);
  mlpParams.initIns();
  mlpParams.initOuts();
  mlpParams.initRefs();
  
  hipError_t result;
  int epochs = 20;
  int warmup = 10;

  if (doChecking) {
    //Run our reference MLP
    result = referenceMLP(mlpParams);
    if (result != hipSuccess) {
      return 1;
    }
  }

  //Run baseline MLP
  double baselineTime = 0;
  double matmul1Time = 0;
  double softmaxTime = 0;
  double matmul2Time = 0;

  if (policy == "baseline") {
  if (mlpParams.isGPT3()) {
    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                             baselineTime, matmul1Time, softmaxTime, matmul2Time, 1);

    CUDA_CHECK(hipDeviceSynchronize());

    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                             baselineTime, matmul1Time, softmaxTime, matmul2Time, warmup);

    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-BASELINE:\n");
    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                         baselineTime, matmul1Time, softmaxTime, matmul2Time, epochs);
    CUDA_CHECK(result);
    printf("END-BASELINE:\n");
    printf("Average time %lf microseconds\n", baselineTime/(float)epochs);
  } else if (mlpParams.isLLaMa()) {
    printf("LLaMa not supported in baseline\n");
  }
  }

  
  if (doChecking) {
    mlpParams.initOuts();
  }
  //Setup cusync gemm
  cutlass::gemm::GemmCoord tileSizeCoord1{ShapeThreadBlock1::kM, ShapeThreadBlock1::kN, 1};
  printf("ShapeThreadBlock1::kM = %d, ShapeThreadBlock1::kN = %d\n", ShapeThreadBlock1::kM, ShapeThreadBlock1::kN);

  cutlass::gemm::GemmCoord tileSizeCoord2{ShapeThreadBlock2::kM, ShapeThreadBlock2::kN, 1};

  cutlass::gemm::GemmCoord gridDim1 = CuSyncGeMMSwizzle().get_tiled_shape(mlpParams.gemm_size1, tileSizeCoord1, split_k1);  // 这里继承自原生cutlass的GemmHorizontalThreadblockSwizzle，输入problem_size和block_size，然后除一下。
  printf("gridDim1: m = %d, n = %d, k = %d\n", gridDim1.m(), gridDim1.n(), gridDim1.k());
  cutlass::gemm::GemmCoord gridDim2 = CuSyncGeMMSwizzle().get_tiled_shape(mlpParams.gemm_size2, tileSizeCoord2, split_k2);

#if defined(ROWSYNC)
  using Sync = RowSync<ShapeThreadBlock1::kM>;
  Sync sync(gridDim1.n());
#elif defined(TILEBATCH)
  using Sync = TileSync<2>;
  Sync sync;
#elif defined(TILESYNC)
  Sync sync;
#elif defined(BATCHEDROW)
  using Sync = BatchedRowSync;
  BatchedRowSync sync(gridDim1.n(), 1);
#else
  #error "Unknown Policy"
#endif

  int highestPriority;
  int lowestPriority;
  CUDA_CHECK(hipDeviceGetStreamPriorityRange(&lowestPriority, &highestPriority));
  CUDA_CHECK(hipStreamCreateWithPriority(&consumer_stream, 0, lowestPriority));
  hipStream_t streams[(lowestPriority - highestPriority + 1)];
  for (int i = highestPriority; i <= lowestPriority; i++) {
    CUDA_CHECK(hipStreamCreateWithPriority(&streams[i - highestPriority], 0, i));
  }
  
  //Run cusync mlp
  if (policy == "cusync") {
  if (mlpParams.isGPT3()) {
    ProdCuStage prod(CuSyncGeMMSwizzle().get_grid_shape(gridDim1), {1,1,1}, NoSync(), sync);
    ConsCuStage cons(CuSyncGeMMSwizzle().get_grid_shape(gridDim2), {1,1,1}, sync, NoSync());

    CuSync::setProducerConsumerPair(prod, cons);
    
    double overlapTime = 0;
    
    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, overlapTime, 1);

    CUDA_CHECK(hipDeviceSynchronize());
    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, overlapTime, warmup);
    
    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-OVERLAPPED:\n");
    
    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, prod, cons, producer_stream, consumer_stream, overlapTime, epochs);
    
    CUDA_CHECK(result);
    printf("END-OVERLAPPED:\n");
    
    printf("Average time %lf microseconds\n", overlapTime/(float)epochs);
  } else if (mlpParams.isLLaMa()) {
    ProdCuStage prod(CuSyncGeMMSwizzle().get_grid_shape(gridDim1), {1,1,1}, NoSync(), sync);
    ConsCuStage cons(CuSyncGeMMSwizzle().get_grid_shape(gridDim2), {1,1,1}, sync, NoSync());
    
    double overlapTime = 0;

    CuSync::setProducerConsumerPair(prod, cons);

    // result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, prod, cons, streams, overlapTime, 1);

    CUDA_CHECK(hipDeviceSynchronize());
    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    // result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, prod, cons, streams, overlapTime, warmup);
    
    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-OVERLAPPED:\n");
    
    // result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, prod, cons, streams, overlapTime, epochs);
    
    CUDA_CHECK(result);
    printf("END-OVERLAPPED:\n");
    
    printf("Average time %lf microseconds\n", overlapTime/(float)epochs);
  }
  }

  return 0;
}
